#include <hip/hip_runtime.h>

static __global__ void transpose(float *in, float *out, uint width) {
    uint tx = blockIdx.x * blockDim.x + threadIdx.x;
    uint ty = blockIdx.y * blockDim.y + threadIdx.y;
    out[tx * width + ty] = in[ty * width + tx];
}

#ifdef __cplusplus
extern "C" {
#endif

void do_transpose(dim3 gDim, dim3 bDim, float *Md, float *Bd, uint WIDTH) {
    transpose<<<gDim, bDim>>>(Md, Bd, WIDTH);
}

void do_transpose2(float *M, uint WIDTH, uint HEIGHT) {
    const int SIZE = WIDTH * HEIGHT * sizeof(float);

    dim3 bDim(3, 3);
    dim3 gDim(WIDTH / bDim.x, HEIGHT / bDim.y);

    float *Md = NULL;
    hipMalloc((void **)&Md, SIZE);
    hipMemcpy(Md, M, SIZE, hipMemcpyHostToDevice);
    float *Bd = NULL;
    hipMalloc((void **)&Bd, SIZE);
    do_transpose(gDim, bDim, Md, Bd, WIDTH);
    hipMemcpy(M, Bd, SIZE, hipMemcpyDeviceToHost);

    // free(M);
}

#ifdef __cplusplus
}
#endif
